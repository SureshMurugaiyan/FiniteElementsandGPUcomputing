#include "hip/hip_runtime.h"
// C++ Script to simulate 2D incompressible flow
#include <iostream>
#include <algorithm>
using namespace std;
#include "input.h"
#include "inputSerial.h"
#include "inputParallel.h"
#include <hip/hip_runtime.h>

// CUDA kernel. Each thread takes care of one element
__global__ void timeStep(double* delt,double* ux,double* uy){
   // Get global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;
if (id < ncGL) {
double umax = 1.0;  // for other cases you need to set this or write function
//double vmax = 1.0;
double C   = 0.9;  // Courant number
double dt1=1;
double dt2=1;
double dt3=1;
double delX = dx;
double dely = dy;
if((dx!=0)&(dy!=0)){ 
 dt1 = C*dx/umax;
 dt2 = C*0.25*Re/((1.0/(delX*delX))+(1.0/(dely*dely)));
 dt3 = C*4/Re;
}

double dtmin = min(dt1,min(dt2,dt3));
//    if (id == 0) {
*delt = dtmin;
//}
}
}
